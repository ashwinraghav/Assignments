#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

//the dimension is hard-coded for readability
#define SIZE 10000

#define BLOCK_SIZE 16
#define ITERATIONS 10000

#define TOP_BOUNDARY_VALUE 0.0
#define BOTTOM_BOUNDARY_VALUE 100.0
#define LEFT_BOUNDARY_VALUE 0.0
#define RIGHT_BOUNDARY_VALUE 100.0
#define INITIAL_CELL_VALUE 50.0

void create_snapshot(float **cells, int n_x, int n_y, int id);
void print_matrix(float**u);
float **allocate_cells(int num_cols, int num_rows);

/*
	kernel funtion
	accepts 2 1D arrays that will be used alternatingly in iterations
*/
__global__ void jacobi(float *x, float *y)
{
	float p, q, r, s, *d_u_new[2];
	int k, next_index=0;
	d_u_new[0] = x;
	d_u_new[1] = y;

	/*Calculate the localized target cells*/
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int i = blockIdx.x*blockDim.x + tx;
	int j = blockIdx.y*blockDim.y + ty;
	int target = i*SIZE+j;
	/*************************************/

	/*
		Each thread copied its corresponding value
 		in global memory to Shared Memory.
		All threads are synced after the copy.
	*/
	__shared__ float shared_cells[BLOCK_SIZE][BLOCK_SIZE][2];
	shared_cells[tx][ty][0] = shared_cells[tx][ty][1] = d_u_new[0][target];
	__syncthreads();
	/*************************************/


	bool is_boundary_cell =  ((target<SIZE)||
				(target%SIZE==0)||
				(target>=SIZE*(SIZE-1))||
				(target%SIZE==(SIZE-1)));
	
	for (k = 0; k < ITERATIONS; k++){
		/*boundary cells are immutable*/
		if(!is_boundary_cell)
		{
			/* if the cell required for relaxation is outside the block
			   pick from global memory, else pick from shared memory*/
			if(tx-1 < 0){
				p = d_u_new[next_index][(i - 1) * SIZE + j];
			}
			else{
				p = shared_cells[tx - 1][ty][next_index];
			}
			if(tx+1 == BLOCK_SIZE){
				q = d_u_new[next_index][(i + 1) * SIZE + j];
			}
			else{
				q = shared_cells[tx + 1][ty][next_index];
			}
			if(ty-1 < 0){
				r = d_u_new[next_index][i * SIZE + j - 1];
			}
			else{
				r = shared_cells[tx][ty - 1][next_index];
			}
			if(ty+1 == BLOCK_SIZE){
				s = d_u_new[next_index][i * SIZE + j + 1];
			}
			else{
				s = shared_cells[tx][ty + 1][next_index];
			}
			/*************************************/
			next_index = (next_index + 1) % 2;
			
			/* 
			   if the computed cell is a boundary 
			   cell, write to shared & global memory
			   else, write only to shared memory
			*/
			if((tx - 1 < 0) || (tx + 1 == BLOCK_SIZE) || (ty - 1 < 0) || (ty + 1 == BLOCK_SIZE)){
				d_u_new[next_index][target] = 0.25 * (p + q + r + s);
			}
			shared_cells[tx][ty][next_index] = 0.25 * (p + q + r + s);
			/*************************************/
			
		}
		/*synchronize after every iteration*/
		__syncthreads();
	}
	/* 
	   After iterations, each thread writes
	   the final computed value back to 
	   global memory for host accessibility
	*/
	if(!truth){
		d_u_new[0][target] = 0.25 * (p + q + r + s);
	}
}

int main()
{
	float *cells[2], *cells_gpu[2], **steady_state;
	int i,j;
	size_t size;
	float h = 1.0/SIZE;
	time_t start_time = time(NULL);

	//The 2D decomposition matrix is flattened
	size=SIZE*SIZE*sizeof(float);
	cells[0]    = (float*)malloc(size);
	cells[1]    = (float*)malloc(size);
	
	//final matrix to generate ppm
	steady_state = allocate_cells(SIZE, SIZE);
	
	//allocate global memory
	hipMalloc(&cells_gpu[0],size);
	hipMalloc(&cells_gpu[1],size);

	//initialize cell values on host before copy*/
	for(i=0;i<SIZE;i++)
	{
		for(j=0;j<SIZE;j++)
		{
			cells[0][i*SIZE+j] = cells[1][i*SIZE+j] = INITIAL_CELL_VALUE;
		}
	}

	for(i=0;i<SIZE;i++)
	{
		cells[0][i] = cells[1][i] = TOP_BOUNDARY_VALUE;
		cells[0][i*SIZE] = cells[1][i*SIZE] = LEFT_BOUNDARY_VALUE;
		cells[0][SIZE*(SIZE-1)+i] = cells[1][SIZE*(SIZE-1)+i] = BOTTOM_BOUNDARY_VALUE;
		cells[0][i*SIZE+SIZE-1] = cells[1][i*SIZE+SIZE-1] = RIGHT_BOUNDARY_VALUE;
	}
	/*************************************/

	//copy decomposition matrix to CUDA global memory*/
	hipMemcpy(cells_gpu[0], cells[0], size, hipMemcpyHostToDevice);
	hipMemcpy(cells_gpu[1], cells[1], size, hipMemcpyHostToDevice);
	/*************************************/

	/* 
	   regulate the dimentions of the block
	   and the grid containing the blocks
	   ****Ugly code omitted******
	   
	*/
	dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
	dim3 dimGrid(SIZE/BLOCK_SIZE,SIZE/BLOCK_SIZE);
	/*************************************/
	
	/*Kernel queuing*/
	jacobi<<<dimGrid,dimBlock>>>(cells_gpu[0], cells_gpu[1]);
	//note that all threads are synchronized internally
	
	/* 
	   copy final values to device from global memory
	   and unflatten the 1D array for ppm generation
	*/
	hipMemcpy(cells[0], cells_gpu[0], size, hipMemcpyDeviceToHost);

	for(i=0;i<SIZE;i++)
	{
		for(j=0; j < SIZE; j++){
			steady_state[i][j] = cells[0][i*SIZE+j];
		}
	}
	create_snapshot(cells, SIZE-2, SIZE-2, ITERATIONS);
	/*************************************/
	
	time_t end_time = time(NULL);
	printf("\nExecution time: %d seconds\n", (int) difftime(end_time, start_time));

	//Free all allocated memory
	free(cells[0]);
	free(cells[1]);
	hipFree(cells_gpu[0]);
	hipFree(cells_gpu[1]);

}

float **allocate_cells(int num_cols, int num_rows) {
	float **array = (float **) malloc(num_rows * sizeof(float *));
	
	array[0] = (float *) malloc(num_rows * num_cols * sizeof(float));

	int i;
	for (i = 1; i < num_rows; i++) {
		array[i] = array[0] + (i * num_cols);
	}

	return array;
}

void create_snapshot(float **cells, int num_cols, int num_rows, int id) {
	//Code omitted
}

//for debugging
void print_matrix(float**u)
{
	//Code omitted
}
