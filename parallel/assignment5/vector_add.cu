#include "hip/hip_runtime.h"
// This CUDA program implements vector addition on both the CPU & GPU
//
// To compile:
//  nvcc -O3 vector_add.cu -o vector_add
// Note: if /usr/local/cuda/bin is not in your $PATH, you will need to
//  specify the full path to nvcc: /usr/local/cuda/bin/nvcc
//
// To run:
//  ./vector_add N
//  (where N is the number of elements in each vector)


#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

// Function declarations
float *CPU_add_vectors(float *A, float *B, int N);
float *GPU_add_vectors(float *A, float *B, int N);
float *get_random_vector(int N);
long long start_timer();
long long stop_timer(long long start_time, char *name);
void die(char *message);
void check_error(hipError_t e);


// The number of threads per blocks in the kernel
// (if we define it here, then we can use its value in the kernel,
//  for example to statically declare an array in shared memory)
const int threads_per_block = 256;


int main(int argc, char **argv) {
	// Seed the random generator (use a constant here for repeatable results)
	srand(4444);

	// Determine the vector length
	int N = 100000;  // default value
	if (argc > 1) N = atoi(argv[1]); // user-specified value

	// Generate two random vectors
	long long vector_start_time = start_timer();
	float *A = get_random_vector(N);
	float *B = get_random_vector(N);
	stop_timer(vector_start_time, "Vector generation");
	
	// Compute their sum on the GPU
	long long GPU_start_time = start_timer();
	float *C_GPU = GPU_add_vectors(A, B, N);
	long long GPU_time = stop_timer(GPU_start_time, "\t            Total");
	
	// Compute their sum on the CPU
	long long CPU_start_time = start_timer();
	float *C_CPU = CPU_add_vectors(A, B, N);
	long long CPU_time = stop_timer(CPU_start_time, "\nCPU");
	
	// Compute the speedup or slowdown
	if (GPU_time > CPU_time) printf("\nCPU outperformed GPU by %.2fx\n", (float) GPU_time / (float) CPU_time);
	else                     printf("\nGPU outperformed CPU by %.2fx\n", (float) CPU_time / (float) GPU_time);
	
	// Check the correctness of the GPU results
	int num_wrong = 0;
	for (int i = 0; i < N; i++) {
		if (fabs(C_CPU[i] - C_GPU[i]) > 0.000001) num_wrong++;
	}
	
	// Report the correctness results
	if (num_wrong) printf("\n%d / %d values incorrect\n", num_wrong, N);
	else           printf("\nAll values correct\n");

}


// A GPU kernel that computes the vector sum A + B
// (each thread computes a single value of the result)
__global__ void add_vectors_kernel(float *A, float *B, float *C, int N) {
	// Determine which element this thread is computing
	// Note: here we need to convert from the two-dimensional grid of
	//  thread blocks back to a one-dimensional index into the arrays
	int block_id = blockIdx.x + gridDim.x * blockIdx.y;
	int thread_id = blockDim.x * block_id + threadIdx.x;
	
	// Make sure we do not go off the end of the array
	if (thread_id < N) {
		// Compute a single element of the result vector
		C[thread_id] = A[thread_id] + B[thread_id];
	}
}


// Returns the vector sum A + B (computed on the GPU)
float *GPU_add_vectors(float *A_CPU, float *B_CPU, int N) {
	
	long long memory_start_time = start_timer();

	// Allocate GPU memory for the inputs and the result
	int vector_size = N * sizeof(float);
	float *A_GPU, *B_GPU, *C_GPU;
	check_error(hipMalloc((void **) &A_GPU, vector_size));
	check_error(hipMalloc((void **) &B_GPU, vector_size));
	check_error(hipMalloc((void **) &C_GPU, vector_size));
	
	// Transfer the input vectors to GPU memory
	check_error(hipMemcpy(A_GPU, A_CPU, vector_size, hipMemcpyHostToDevice));
	check_error(hipMemcpy(B_GPU, B_CPU, vector_size, hipMemcpyHostToDevice));
	
	stop_timer(memory_start_time, "\nGPU:\t  Transfer to GPU");
	
	// Determine the number of thread blocks in the x- and y-dimension
	// Note: we use a two-dimensional grid of thread blocks here because each dimension
	//  of the grid can only have up to 64K thread blocks; if we want to use more than
	//  64K thread blocks, we need to use a two-dimensional grid. This is slightly
	//  awkward, however, since the underlying problem is inherently one-dimensional
	int num_blocks = (N + threads_per_block - 1) / threads_per_block;
	int max_blocks_per_dimension = 65535;
	int num_blocks_y = (int) ((float) (num_blocks + max_blocks_per_dimension - 1) / (float) max_blocks_per_dimension);
	int num_blocks_x = (int) ((float) (num_blocks + num_blocks_y - 1) / (float) num_blocks_y);
	dim3 grid_size(num_blocks_x, num_blocks_y, 1);
	
	// Execute the kernel to compute the vector sum on the GPU
	long long kernel_start_time = start_timer();
	add_vectors_kernel <<< grid_size , threads_per_block >>> (A_GPU, B_GPU, C_GPU, N);
	hipDeviceSynchronize();  // This is only needed for timing and error-checking purposes
	stop_timer(kernel_start_time, "\t Kernel execution");
	
	// Check for kernel errors
	check_error(hipGetLastError());
	
	// Allocate CPU memory for the result
	float *C_CPU = (float *) malloc(vector_size);
	if (C_CPU == NULL) die("Error allocating CPU memory");
	
	// Transfer the result from the GPU to the CPU
	memory_start_time = start_timer();
	check_error(hipMemcpy(C_CPU, C_GPU, vector_size, hipMemcpyDeviceToHost));
	stop_timer(memory_start_time, "\tTransfer from GPU");
	
	// Free the GPU memory
	check_error(hipFree(A_GPU));
	check_error(hipFree(B_GPU));
	check_error(hipFree(C_GPU));
	
	return C_CPU;
}


// Returns the vector sum A + B
float *CPU_add_vectors(float *A, float *B, int N) {	
	// Allocate memory for the result
	float *C = (float *) malloc(N * sizeof(float));
	if (C == NULL) die("Error allocating CPU memory");

	// Compute the sum;
	for (int i = 0; i < N; i++) C[i] = A[i] + B[i];
	
	// Return the result
	return C;
}


// Returns a randomized vector containing N elements
float *get_random_vector(int N) {
	if (N < 1) die("Number of elements must be greater than zero");
	
	// Allocate memory for the vector
	float *V = (float *) malloc(N * sizeof(float));
	if (V == NULL) die("Error allocating CPU memory");
	
	// Populate the vector with random numbers
	for (int i = 0; i < N; i++) V[i] = (float) rand() / (float) rand();
	
	// Return the randomized vector
	return V;
}


// Returns the current time in microseconds
long long start_timer() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec * 1000000 + tv.tv_usec;
}


// Prints the time elapsed since the specified time
long long stop_timer(long long start_time, char *label) {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	long long end_time = tv.tv_sec * 1000000 + tv.tv_usec;
	printf("%s: %.5f sec\n", label, ((float) (end_time - start_time)) / (1000 * 1000));
	return end_time - start_time;
}


// Prints the specified message and quits
void die(char *message) {
	printf("%s\n", message);
	exit(1);
}

// If the specified error code refers to a real error, report it and quit the program
void check_error(hipError_t e) {
	if (e != hipSuccess) {
		printf("\nCUDA error: %s\n", hipGetErrorString(e));
		exit(1);
	}
}
