#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#define SIZE 8192
#define BLOCK_SIZE 16
#define ITERATIONS 10000

#define TOP_BOUNDARY_VALUE 0.0
#define BOTTOM_BOUNDARY_VALUE 100.0
#define LEFT_BOUNDARY_VALUE 0.0
#define RIGHT_BOUNDARY_VALUE 100.0
#define INITIAL_CELL_VALUE 50.0

void create_snapshot(float **cells, int n_x, int n_y, int id);

float ratio(float**u,float ant,int iter)
{
	float tmp=0.0;
	int i,j;
	for(i=0;i<SIZE;i++)
	{
		for(j=0;j<SIZE;j++)
		{
			if(u[i][j]>tmp)
				tmp=u[i][j];
		}
	}
	if(iter%10==0)
		printf(" iter=%d ratio=%f max=%f\n",iter,tmp/ant,tmp);
	return tmp;
}

void print_matrix(float**u)
{
	int i,j;
	for(i=0;i<SIZE;i++)
	{
		for(j=0;j<SIZE;j++)
		{
			printf("%f ",u[i][j]);
		}
		printf("\n");
	}
}
__global__ void jacobi(float *d_u_new,float *d_u)
{
	float p, q, r, s;
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int i = blockIdx.x*blockDim.x + tx;
	int j = blockIdx.y*blockDim.y + ty;

	int target = i*SIZE+j;
	__shared__ float shared_cells[BLOCK_SIZE][BLOCK_SIZE];
	
	shared_cells[tx][ty] = d_u[target];
	__syncthreads(); 
	
	if((target<SIZE)||(target%SIZE==0)||(target>=SIZE*(SIZE-1))||(target%SIZE==(SIZE-1))){
	}
	else
	{
		if(tx-1 < 0){
			p = d_u[(i - 1) * SIZE + j];
		}
		else{
			p = shared_cells[tx - 1][ty];
		}
		if(tx+1 == BLOCK_SIZE){
			q = d_u[(i + 1) * SIZE + j];
		}
		else{
			q = shared_cells[tx + 1][ty];
		}
		if(ty-1 < 0){
			r = d_u[i * SIZE + j - 1];
		}
		else{
			r = shared_cells[tx][ty - 1];
		}
		if(ty+1 == BLOCK_SIZE){
			s = d_u[i * SIZE + j + 1];
		}
		else{
			s = shared_cells[tx][ty + 1];
		}
		d_u_new[target] = 0.25 * (p + q + r + s);
		//d_u_new[target]=0.25*( p + shared_cells[tx + 1][ty] + shared_cells[tx][ty - 1] + shared_cells[tx][ty + 1]);
	}
	if((tx > 0) && (tx < BLOCK_SIZE - 1) && (ty > 0) && (ty < BLOCK_SIZE - 1)){
	}
}

float **allocate_cells(int num_cols, int num_rows) {
	float **array = (float **) malloc(num_rows * sizeof(float *));
	
	array[0] = (float *) malloc(num_rows * num_cols * sizeof(float));

	int i;
	for (i = 1; i < num_rows; i++) {
		array[i] = array[0] + (i * num_cols);
	}

	return array;
}

// Sets all of the specified cells to their initial value.
int main()
{
	float *cells[2], *cells_gpu[2], **steady_state;
	int i,j;
	size_t size;
	float h = 1.0/SIZE;


	size=SIZE*SIZE*sizeof(float);
	printf("Necesitamos %d Mb\n",3*size/1024/1024);
	cells[0]    = (float*)malloc(size);
	cells[1]    = (float*)malloc(size);
	
	steady_state = allocate_cells(SIZE, SIZE);
	
	hipMalloc(&cells_gpu[0],size);
	hipMalloc(&cells_gpu[1],size);


	for(i=0;i<SIZE;i++)
	{
		for(j=0;j<SIZE;j++)
		{
			cells[0][i*SIZE+j] = cells[1][i*SIZE+j] = INITIAL_CELL_VALUE;
		}
	}

	for(i=0;i<SIZE;i++)
	{
		cells[0][i] = cells[1][i] = TOP_BOUNDARY_VALUE;
		cells[0][i*SIZE] = cells[1][i*SIZE] = LEFT_BOUNDARY_VALUE;
		cells[0][SIZE*(SIZE-1)+i] = cells[1][SIZE*(SIZE-1)+i] = BOTTOM_BOUNDARY_VALUE;
		cells[0][i*SIZE+SIZE-1] = cells[1][i*SIZE+SIZE-1] = RIGHT_BOUNDARY_VALUE;
	}

	hipMemcpy(cells_gpu[0], cells[0], size, hipMemcpyHostToDevice);
	hipMemcpy(cells_gpu[1], cells[1], size, hipMemcpyHostToDevice);

	dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
	dim3 dimGrid(SIZE/BLOCK_SIZE,SIZE/BLOCK_SIZE);
	
	time_t start_time = time(NULL);
	for(i=0;i<ITERATIONS;i++)
	{
		if(i%2==0)
			jacobi<<<dimGrid,dimBlock>>>(cells_gpu[0], cells_gpu[1]);
		else
			jacobi<<<dimGrid,dimBlock>>>(cells_gpu[1], cells_gpu[0]);
		if(i%10==0)
			printf("iter=%d\n",i);
	}

	int final_cells = (ITERATIONS % 2 == 0) ? 1 : 0;
	hipMemcpy(cells[0], cells_gpu[final_cells], size, hipMemcpyDeviceToHost);

	for(i=0;i<SIZE;i++)
	{
		for(j=0; j < SIZE; j++){
			steady_state[i][j] = cells[0][i*SIZE+j];
		//	printf("%f ", cells[0][i*SIZE+j]);
		}
		//printf("\n");
	}
	time_t end_time = time(NULL);
	printf("\nExecution time: %d seconds\n", (int) difftime(end_time, start_time));
	//create_snapshot(cells, SIZE-2, SIZE-2, ITERATIONS);

	/* Liberamos memoria */
	free(cells[0]);
	free(cells[1]);
	hipFree(cells_gpu[0]);
	hipFree(cells_gpu[1]);

}
void create_snapshot(float **cells, int num_cols, int num_rows, int id) {
	int scale_x, scale_y;
	scale_x = scale_y = 1;
	
	// Figure out if we need to scale down the snapshot (to 1,000 x 1,000)
	//  and, if so, how much to scale down
	if (num_cols > 1000) {
		if ((num_cols % 1000) == 0) scale_x = num_cols / 1000;
		else {
			return;
		}
	}
	if (num_rows > 1000) {
		if ((num_rows % 1000) == 0) scale_y = num_rows / 1000;
		else {
			printf("Cannot create snapshot for y-dimensions >1,000 that are not multiples of 1,000!\n");
			return;
		}
	}
	
	// Open/create the file
	char text[255];
	sprintf(text, "snapshot.%d.ppm", id);
	FILE *out = fopen(text, "w");
	// Make sure the file was created
	if (out == NULL) {
		printf("Error creating snapshot file!\n");
		return;
	}
	
	// Write header information to file
	// P3 = RGB values in decimal (P6 = RGB values in binary)
	fprintf(out, "P3 %d %d 100\n", num_cols / scale_x, num_rows / scale_y);
	
	// Precompute the value needed to scale down the cells
	float inverse_cells_per_pixel = 1.0 / ((float) scale_x * scale_y);
	
	// Write the values of the cells to the file
	int x, y, i, j;
	for (y = 1; y <= num_rows; y += scale_y) {
		for (x = 1; x <= num_cols; x += scale_x) {
			float sum = 0.0;
			for (j = y; j < y + scale_y; j++) {
				for (i = x; i < x + scale_x; i++) {
					sum += cells[j][i];
				}
			}
			// Write out the average value of the cells we just visited
			int average = (int) (sum * inverse_cells_per_pixel);
			fprintf(out, "%d 0 %d\t", average, 100 - average);
		}
		fwrite("\n", sizeof(char), 1, out);
	}
	
	// Close the file
	fclose(out);
}
