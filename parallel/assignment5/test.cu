#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#define SIZE 8092
#define BLOCK_SIZE 16
#define ITERATIONS 10000

#define TOP_BOUNDARY_VALUE 0.0
#define BOTTOM_BOUNDARY_VALUE 100.0
#define LEFT_BOUNDARY_VALUE 0.0
#define RIGHT_BOUNDARY_VALUE 100.0
#define INITIAL_CELL_VALUE 50.0

void create_snapshot(float **cells, int n_x, int n_y, int id);

float ratio(float**u,float ant,int iter)
{
	float tmp=0.0;
	int i,j;
	for(i=0;i<SIZE;i++)
	{
		for(j=0;j<SIZE;j++)
		{
			if(u[i][j]>tmp)
				tmp=u[i][j];
		}
	}
	if(iter%10==0)
		printf(" iter=%d ratio=%f max=%f\n",iter,tmp/ant,tmp);
	return tmp;
}

void print_matrix(float**u)
{
	int i,j;
	for(i=0;i<SIZE;i++)
	{
		for(j=0;j<SIZE;j++)
		{
			printf("%f ",u[i][j]);
		}
		printf("\n");
	}
}
__global__ void jacobi(float *x, float *y, int iteration)
{
	float p, q, r, s, *d_u_new[2];
	d_u_new[0] = x;
	d_u_new[1] = y;

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int i = blockIdx.x*blockDim.x + tx;
	int j = blockIdx.y*blockDim.y + ty;
	int k, next_index=0;

	int target = i*SIZE+j;

	__shared__ float shared_cells[BLOCK_SIZE][BLOCK_SIZE][2];
	shared_cells[tx][ty][0] = shared_cells[tx][ty][1] = d_u_new[0][target];
	__syncthreads();

	bool truth =  ((target<SIZE)||(target%SIZE==0)||(target>=SIZE*(SIZE-1))||(target%SIZE==(SIZE-1)));
	next_index = (iteration) % 2;

	if(!truth)
	{
		if(tx-1 < 0){
			p = d_u_new[next_index][(i - 1) * SIZE + j];
		}
		else{
			p = shared_cells[tx - 1][ty][next_index];
		}
		if(tx+1 == BLOCK_SIZE){
			q = d_u_new[next_index][(i + 1) * SIZE + j];
		}
		else{
			q = shared_cells[tx + 1][ty][next_index];
		}
		if(ty-1 < 0){
			r = d_u_new[next_index][i * SIZE + j - 1];
		}
		else{
			r = shared_cells[tx][ty - 1][next_index];
		}
		if(ty+1 == BLOCK_SIZE){
			s = d_u_new[next_index][i * SIZE + j + 1];
		}
		else{
			s = shared_cells[tx][ty + 1][next_index];
		}
		if((tx - 1 < 0) || (tx + 1 == BLOCK_SIZE) || (ty - 1 < 0) || (ty + 1 == BLOCK_SIZE)){
			d_u_new[next_index][target] = 0.25 * (p + q + r + s);
		}else{
			shared_cells[tx][ty][next_index] = 0.25 * (p + q + r + s);
		}

		//__syncthreads();
	}
	if (iteration == ITERATIONS-1){
		if(!truth){
			d_u_new[0][target] = 0.25 * (p + q + r + s);
		}
	}
}


float **allocate_cells(int num_cols, int num_rows) {
	float **array = (float **) malloc(num_rows * sizeof(float *));
	
	array[0] = (float *) malloc(num_rows * num_cols * sizeof(float));

	int i;
	for (i = 1; i < num_rows; i++) {
		array[i] = array[0] + (i * num_cols);
	}

	return array;
}

// Sets all of the specified cells to their initial value.
int main()
{
	float *cells[2], *cells_gpu[2], **steady_state;
	int i,j;
	size_t size;
	float h = 1.0/SIZE;


	size=SIZE*SIZE*sizeof(float);
	cells[0]    = (float*)malloc(size);
	cells[1]    = (float*)malloc(size);
	
	steady_state = allocate_cells(SIZE, SIZE);
	
	hipMalloc(&cells_gpu[0],size);
	hipMalloc(&cells_gpu[1],size);


	for(i=0;i<SIZE;i++)
	{
		for(j=0;j<SIZE;j++)
		{
			cells[0][i*SIZE+j] = cells[1][i*SIZE+j] = INITIAL_CELL_VALUE;
		}
	}

	for(i=0;i<SIZE;i++)
	{
		cells[0][i] = cells[1][i] = TOP_BOUNDARY_VALUE;
		cells[0][i*SIZE] = cells[1][i*SIZE] = LEFT_BOUNDARY_VALUE;
		cells[0][SIZE*(SIZE-1)+i] = cells[1][SIZE*(SIZE-1)+i] = BOTTOM_BOUNDARY_VALUE;
		cells[0][i*SIZE+SIZE-1] = cells[1][i*SIZE+SIZE-1] = RIGHT_BOUNDARY_VALUE;
	}

	hipMemcpy(cells_gpu[0], cells[0], size, hipMemcpyHostToDevice);
	hipMemcpy(cells_gpu[1], cells[1], size, hipMemcpyHostToDevice);

	dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
	dim3 dimGrid(SIZE/BLOCK_SIZE,SIZE/BLOCK_SIZE);
	
	time_t start_time = time(NULL);
	int k;
	for (k = 0; k < ITERATIONS; k++)
	{
		jacobi<<<dimGrid,dimBlock>>>(cells_gpu[0], cells_gpu[1], k);
	}
	hipMemcpy(cells[0], cells_gpu[0], size, hipMemcpyDeviceToHost);
	time_t end_time = time(NULL);

	for(i=0;i<SIZE;i++)
	{
		for(j=0; j < SIZE; j++){
			steady_state[i][j] = cells[0][i*SIZE+j];
		//	printf("%f ", cells[0][i*SIZE+j]);
		}
	//	printf("\n");
	}
	printf("\nExecution time: %d seconds\n", (int) difftime(end_time, start_time));
	create_snapshot(steady_state, SIZE-2, SIZE-2, ITERATIONS);

	/* Liberamos memoria */
	free(cells[0]);
	free(cells[1]);
	hipFree(cells_gpu[0]);
	hipFree(cells_gpu[1]);

}
void create_snapshot(float **cells, int num_cols, int num_rows, int id) {
	int scale_x, scale_y;
	scale_x = scale_y = 1;
	
	// Figure out if we need to scale down the snapshot (to 1,000 x 1,000)
	//  and, if so, how much to scale down
	if (num_cols > 1000) {
		if ((num_cols % 1000) == 0) scale_x = num_cols / 1000;
		else {
			return;
		}
	}
	if (num_rows > 1000) {
		if ((num_rows % 1000) == 0) scale_y = num_rows / 1000;
		else {
			printf("Cannot create snapshot for y-dimensions >1,000 that are not multiples of 1,000!\n");
			return;
		}
	}
	
	// Open/create the file
	char text[255];
	sprintf(text, "snapshot.%d.ppm", id);
	FILE *out = fopen(text, "w");
	// Make sure the file was created
	if (out == NULL) {
		printf("Error creating snapshot file!\n");
		return;
	}
	
	// Write header information to file
	// P3 = RGB values in decimal (P6 = RGB values in binary)
	fprintf(out, "P3 %d %d 100\n", num_cols / scale_x, num_rows / scale_y);
	
	// Precompute the value needed to scale down the cells
	float inverse_cells_per_pixel = 1.0 / ((float) scale_x * scale_y);
	
	// Write the values of the cells to the file
	int x, y, i, j;
	for (y = 1; y <= num_rows; y += scale_y) {
		for (x = 1; x <= num_cols; x += scale_x) {
			float sum = 0.0;
			for (j = y; j < y + scale_y; j++) {
				for (i = x; i < x + scale_x; i++) {
					sum += cells[j][i];
				}
			}
			// Write out the average value of the cells we just visited
			int average = (int) (sum * inverse_cells_per_pixel);
			fprintf(out, "%d 0 %d\t", average, 100 - average);
		}
		fwrite("\n", sizeof(char), 1, out);
	}
	
	// Close the file
	fclose(out);
}
