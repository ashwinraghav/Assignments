#include "hip/hip_runtime.h"
#ifndef _HOTPLATE_KERNEL_H_
#define _HOTPLATE_KERNEL_H_
// The width of external cells that surround the plate's cells
#define BOUNDARYCELLS 2
#define THREAD_BLOCK_WIDTH 128
#define THREAD_BLOCK_HEIGHT 4
#define FOR(A) for(int A=1;A<=BOUNDARYCELLS;A++)
#define SYNCHRONIZATION __syncthreads()
#define NN_MEAN(array,a,b) (array[a][b+1]+array[a+1][b]+array[a-1][b]+array[a][b-1])*0.25
#define GHOST 4
//Used several macros as opposed to functions to provide opportunities for vectorization
#define GRID_BOTTOM_BLOCK (blockIdx.y==(gridDim.y-1))
#define LEFT_THREAD (!threadIdx.x)
#define TOP_THREAD (!threadIdx.y)
#define RIGHT_THREAD(threadIdx.x==(THREAD_BLOCK_WIDTH-1))
#define BOTTOM_THREAD(threadIdx.y==(THREAD_BLOCK_HEIGHT-1)) 

#define COPY(A,B,C) shared_cells[A][B]=gplate_block_cellsInput[C];

void create_snapshot(float **cells, int n_x, int n_y, int id);
__global__ void Kernel(
		dim3 plateDims,
		float* g_plateblock_cellsInput,
		float* g_plateblock_cellsOutput, dim3 gridDims)
{
	//Executing thread's location
	dim3 myCell((blockIdx.x*blockDim.x)+threadIdx.x+BOUNDARYCELLS,
			(blockIdx.y*blockDim.y)+threadIdx.y+BOUNDARYCELLS);
	if ((myCell.x >= plateDims.x-BOUNDARYCELLS) || (myCell.y >= plateDims.y - BOUNDARYCELLS)) {
		// My location is outside of the plate boundaries
		return;
	}
	/*calculating positions with respect to 1D array and thread position on the grid*/
	int left=myCell.x-1;
	int right=myCell.x+1;
	int top=myCell.y-1;
	int bottom=myCell.y+1;
	int me=myCell.x;
	int bc2=BOUNDARYCELLS;
	int x_offset_in_block=threadIdx.x+BOUNDARYCELLS; //column
	int y_offset_in_block=threadIdx.y+BOUNDARYCELLS; //row
	bc2=bc2<<1;
	float res=0.0f;
	__shared__ float shared_cells[THREAD_BLOCK_HEIGHT+(2*BOUNDARYCELLS)][THREAD_BLOCK_WIDTH+(BOUNDARYCELLS*2)]; 

	shared_cells[y_offset_in_block][x_offset_in_block]=g_plateblock_cellsInput[myCell.y*plateDims.x+myCell.x]; //I put my value first
	//Top thread gets the required cells at the top
	if(TOP_THREAD) 
	{
		FOR(i)
		{
			top=myCell.y-i;
			int temp=y_offset_in_block-i;
			COPY(temp, xoffset_in_block, top*plateDims.x+me)
		}
		//Left thread gets ghost cells      
		if(LEFT_THREAD) 
		{
			FOR(i)
			{
				top=myCell.y-i;
				int temp=y_offset_in_block-i;
				FOR(j)
				{
					left=myCell.x-j;
					COPY(temp, xoffset_in_block, top*plateDims.x+left)
				}
			}
		}
		//unless the executing thread handles the right block
		if(!GRID_RIGHT_BLOCK) 
		{
			//Right Threads get ghost cells
			if(RIGHT_THREAD)
			{
				FOR(i)
				{
					top=myCell.y-i;
					int temp=y_offset_in_block-i;
					FOR(j)
					{
						right=myCell.x+j;
						COPY(temp, xoffset_in_block, top*plateDims.x+right)
					}
				}
			}
		}
 		//Executing thread handles grid's right block
		else
		{
			//Last thread in the right block
			if(threadIdx.x==(plateDims.x-(bc2)-1)%THREAD_BLOCK_WIDTH) 
			{
				FOR(i)
				{
					top=myCell.y-i;
					int temp=y_offset_in_block-i;
					FOR(j)
					{
						right=myCell.x+j;
						COPY(temp, xoffset_in_block + j, top*plateDims.x+right)
					}   
				}
			}
		}
	}

	
	if(!GRID_BOTTOM_BLOCK)
	{
		//bottom thread gets required bottom cells
		if(BOTTOM_THREAD)
		{
			FOR(i)
			{
				int temp=y_offset_in_block+i;
				bottom=myCell.y+i;
				COPY(temp, xoffset_in_block + j, bottom*plateDims.x+me)
			}
			//get ghost cells 
			if(LEFT_THREAD) 
			{
				FOR(i)
				{
					bottom=myCell.y+i;
					FOR(j)
					{
						left=myCell.x-j;
						COPY(y_offset_in_block+i, xoffset_in_block - j, bottom*plateDims.x+left)
					}
				}
			}
			if(GRID_RIGHT_BLOCK)
			{
				if(RIGHT_THREAD)
				{
					FOR(i)
					{
						bottom=myCell.y+i;
						int temp=y_offset_in_block+i;
						FOR(j)
						{
							COPY(temp, x_offset_in_block+j, bottom*plateDims.x+right)
						}
					}
				}
			}
			else 
			{
				if(threadIdx.x==(plateDims.x-bc2-1)%THREAD_BLOCK_WIDTH)
				{
					FOR(i)
					{
						bottom=myCell.y+i;
						FOR(j)
						{
							right=myCell.x+j;
							COPY(y_offset_in_block+i, x_offset_in_block+j, bottom*plateDims.x+right)
						}
					}
				}
			}
			/***********************/
		}
	}
	else
	{
		if(threadIdx.y==(plateDims.y-(bc2)-1)%THREAD_BLOCK_HEIGHT)
		{
			FOR(i)
				shared_cells[y_offset_in_block+i][x_offset_in_block]=g_plateblock_cellsInput[(myCell.y+i)*plateDims.x +me];
			if(LEFT_THREAD)
			{
				FOR(i)
				{
					bottom=myCell.y+i;
					FOR(j)    
					{
						left=myCell.x-j;
						COPY(y_offset_in_block+i, x_offset_in_block-j, bottom*plateDims.x+left)
					}
				}
			}
			if(!GRID_RIGHT_BLOCK) 
			{
				if(RIGHT_THREAD)
				{
					FOR(i)
					{
						bottom=myCell.y+i;
						FOR(j)
						{
							right=myCell.x+j;
							COPY(y_offset_in_block+i, x_offset_in_block+j, bottom*plateDims.x+right)
						}
					}
				}
			}
			else
			{
				if(threadIdx.x==(plateDims.x-bc2-1)%THREAD_BLOCK_WIDTH) 
				{
					FOR(i)
					{
						bottom=myCell.y+i;
						FOR(j)
							COPY(y_offset_in_block+i, x_offset_in_block+j, bottom*plateDims.x+myCell.x+j)
					}
				}
			}
		}
	}
	if(LEFT_THREAD) 
	{
		FOR(i)
			COPY(y_offset_in_block, x_offset_in_block-i, myCell.y*plateDims.x+myCell.x-i)
	}
	if(!GRID_RIGHT_BLOCK)
	{
		if(RIGHT_THREAD)
		{
			FOR(i)

				COPY(y_offset_in_block, x_offset_in_block+i, myCell.y*plateDims.x+myCell.x+i)
		}
	}
	else
	{          if(threadIdx.x==(plateDims.x-bc2-1)%THREAD_BLOCK_WIDTH) 
		{
			FOR(i)

				COPY(y_offset_in_block, x_offset_in_block+i, myCell.y*plateDims.x+myCell.x+i)
		}
	}
	SYNCHRONIZATION;
	res=NN_MEAN(shared_cells,y_offset_in_block,x_offset_in_block);
	
	//Move all the ghost cells to iteration i+1
	float abv_new_state=0.0f,blw_new_state=0.0f,left_new_state=0.0f,right_new_state=0.0f;
	float abv_new_state_array[BOUNDARYCELLS];
	
	//Hacky. Boundary blocks should not update ghosts since cells are immutable
	if(TOP_THREAD)
	{
		//update ghosts too
		if(!blockIdx.y) 
		{
			for(int i=1;i<BOUNDARYCELLS;i++)
				abv_new_state_array[i-1]=shared_cells[y_offset_in_block-i][x_offset_in_block]; 
		}
		//The first block on the grid should not update ghost cells !
		else  
		{
			for(int i=1;i<BOUNDARYCELLS;i++)
				abv_new_state_array[i-1]=(shared_cells[y_offset_in_block-i-1][x_offset_in_block] +
						shared_cells[y_offset_in_block-i][x_offset_in_block-i] +
						shared_cells[y_offset_in_block-i][x_offset_in_block+i] +
						shared_cells[y_offset_in_block][x_offset_in_block])*0.25f;
		}
		abv_new_state=abv_new_state_array[0];
	}
	if(LEFT_THREAD)
	{
		if(!blockIdx.x) 
			left_new_state=shared_cells[y_offset_in_block][x_offset_in_block-1];

		else
			left_new_state=(shared_cells[y_offset_in_block-1][x_offset_in_block-1] +
					shared_cells[y_offset_in_block][x_offset_in_block-2] +
					shared_cells[y_offset_in_block][x_offset_in_block]+
					shared_cells[1+y_offset_in_block][x_offset_in_block-1])*0.25f;

	}
	if(!GRID_RIGHT_BLOCK)
	{
		if(RIGHT_THREAD) 
			right_new_state=(shared_cells[y_offset_in_block-1][1+x_offset_in_block] +
					shared_cells[y_offset_in_block][x_offset_in_block]+
					shared_cells[y_offset_in_block][x_offset_in_block+2] +
					shared_cells[1+y_offset_in_block][1+x_offset_in_block])*0.25f;

	}
	else
	{
		if(threadIdx.x==(plateDims.x-bc2-1)%THREAD_BLOCK_WIDTH) 
			right_new_state=shared_cells[y_offset_in_block][1+x_offset_in_block];

	}
	if(!GRID_BOTTOM_BLOCK)
	{
		if(BOTTOM_THREAD) 
		{
			blw_new_state=(shared_cells[y_offset_in_block][x_offset_in_block]+
					shared_cells[1+y_offset_in_block][x_offset_in_block-1]+
					shared_cells[1+y_offset_in_block][1+x_offset_in_block] +
					shared_cells[2+y_offset_in_block][x_offset_in_block])*0.25f;
		}
	}
	else
	{
		if(threadIdx.y==(plateDims.y-bc2-1)%THREAD_BLOCK_HEIGHT) //gives the last thread
			blw_new_state=shared_cells[1+y_offset_in_block][x_offset_in_block];

	}
	/****************Relaxation*****************************/
	SYNCHRONIZATION;
	shared_cells[y_offset_in_block][x_offset_in_block]=res;
	// All boundary cells update state !
	if(TOP_THREAD)
		shared_cells[y_offset_in_block-1][x_offset_in_block]=abv_new_state; 
	if(LEFT_THREAD)
		shared_cells[y_offset_in_block][x_offset_in_block-1]=left_new_state;
	if(!GRID_RIGHT_BLOCK)
	{
		if(RIGHT_THREAD)
			shared_cells[y_offset_in_block][1+x_offset_in_block]=right_new_state;
	}
	else
	{
		if(threadIdx.x==(plateDims.x-(bc2)-1)%THREAD_BLOCK_WIDTH)
			shared_cells[y_offset_in_block][1+x_offset_in_block]=right_new_state;
	}
	if(!GRID_BOTTOM_BLOCK) {
		if(BOTTOM_THREAD)
			shared_cells[1+y_offset_in_block][x_offset_in_block]=blw_new_state;
	}
	else 
	{
		if(threadIdx.y==(plateDims.y-(bc2)-1)%THREAD_BLOCK_HEIGHT)
			shared_cells[1+y_offset_in_block][x_offset_in_block]=blw_new_state;
	}

	SYNCHRONIZATION;
	res=NN_MEAN(shared_cells,y_offset_in_block,x_offset_in_block);
	g_plateblock_cellsOutput[myCell.y*plateDims.x+me]=res;
}
#endif 

//debug function
void print_matrix(float**u)
{
	int i,j;
	for(i=0;i<SIZE;i++)
	{
		for(j=0;j<SIZE;j++)
		{
			printf("%f ",u[i][j]);
		}
		printf("\n");
	}
}

float **allocate_cells(int num_cols, int num_rows) {
	float **array = (float **) malloc(num_rows * sizeof(float *));
	
	array[0] = (float *) malloc(num_rows * num_cols * sizeof(float));

	int i;
	for (i = 1; i < num_rows; i++) {
		array[i] = array[0] + (i * num_cols);
	}

	return array;
}

// Sets all of the specified cells to their initial value.
int main()
{
	float *cells[2], *cells_gpu[2], **steady_state;
	int i,j;
	size_t size;
	float h = 1.0/SIZE;


	size=SIZE*SIZE*sizeof(float);
	printf("Necesitamos %d Mb\n",3*size/1024/1024);
	cells[0]    = (float*)malloc(size);
	cells[1]    = (float*)malloc(size);
	
	steady_state = allocate_cells(SIZE, SIZE);
	//Allocate GPU memory
	hipMalloc(&cells_gpu[0],size);
	hipMalloc(&cells_gpu[1],size);


	for(i=0;i<SIZE;i++)
	{
		for(j=0;j<SIZE;j++)
		{
			cells[0][i*SIZE+j] = cells[1][i*SIZE+j] = INITIAL_CELL_VALUE;
		}
	}

	for(i=0;i<SIZE;i++)
	{
		cells[0][i] = cells[1][i] = TOP_BOUNDARY_VALUE;
		cells[0][i*SIZE] = cells[1][i*SIZE] = LEFT_BOUNDARY_VALUE;
		cells[0][SIZE*(SIZE-1)+i] = cells[1][SIZE*(SIZE-1)+i] = BOTTOM_BOUNDARY_VALUE;
		cells[0][i*SIZE+SIZE-1] = cells[1][i*SIZE+SIZE-1] = RIGHT_BOUNDARY_VALUE;
	}

	hipMemcpy(cells_gpu[0], cells[0], size, hipMemcpyHostToDevice);
	hipMemcpy(cells_gpu[1], cells[1], size, hipMemcpyHostToDevice);

	dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
	dim3 dimGrid(SIZE/BLOCK_SIZE,SIZE/BLOCK_SIZE);
	
	time_t start_time = time(NULL);
	//perform relaxation
	for(i=0;i<ITERATIONS/GHOST;i++)
	{
		if(i%2==0)
			jacobi<<<dimGrid,dimBlock>>>(dimBlock, cells_gpu[0], cells_gpu[1], dimGrid);
		else
			jacobi<<<dimGrid,dimBlock>>>(dimBlock, cells_gpu[1], cells_gpu[0], dimGrid);
		if(i%10==0)
			printf("iter=%d\n",i);
	}

	int final_cells = (ITERATIONS % 2 == 0) ? 1 : 0;
	hipMemcpy(cells[0], cells_gpu[final_cells], size, hipMemcpyDeviceToHost);

	//copy the 1D cells to a 2D array for ppm conversion
	for(i=0;i<SIZE;i++)
	{
		for(j=0; j < SIZE; j++){
			steady_state[i][j] = cells[0][i*SIZE+j];
		//	printf("%f ", cells[0][i*SIZE+j]);
		}
		//printf("\n");
	}
	time_t end_time = time(NULL);
	printf("\nExecution time: %d seconds\n", (int) difftime(end_time, start_time));
	//create_snapshot(cells, SIZE-2, SIZE-2, ITERATIONS);

	/* Liberamos memoria */
	free(cells[0]);
	free(cells[1]);
	hipFree(cells_gpu[0]);
	hipFree(cells_gpu[1]);

}
void create_snapshot(float **cells, int num_cols, int num_rows, int id) {
	/*code omitted*/
}
